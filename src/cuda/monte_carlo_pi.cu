#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_SAMPLES 100000000
#define BLOCK_SIZE 256

// Kernel to generate random points and count those inside the circle
__global__ void monte_carlo_pi(int *count, int n, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Create random number generator state
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // Generate random point in [0,1) x [0,1)
    float x = hiprand_uniform(&state);
    float y = hiprand_uniform(&state);

    // Check if point is inside circle
    if (x*x + y*y <= 1.0f) {
        atomicAdd(count, 1);
    }
}

int main() {
    int *d_count;
    int h_count = 0;
    int num_blocks = (NUM_SAMPLES + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Allocate device memory for counter
    hipMalloc(&d_count, sizeof(int));
    hipMemset(d_count, 0, sizeof(int));

    // Launch kernel
    monte_carlo_pi<<<num_blocks, BLOCK_SIZE>>>(d_count, NUM_SAMPLES, time(NULL));

    // Copy result back to host
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    // Calculate pi
    float pi = 4.0f * h_count / NUM_SAMPLES;
    printf("Estimated value of π: %f\n", pi);
    printf("Samples used: %d\n", NUM_SAMPLES);
    printf("Points inside circle: %d\n", h_count);

    // Cleanup
    hipFree(d_count);

    return 0;
}
